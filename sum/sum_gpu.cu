//
// Created by Administrator on 2022/3/8 0008.
//

#include"hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>
#include<sys/time.h>
using namespace std;

__global__ void Plus(float A[],float B[],float C[],int n){
    int i=blockDim.x*blockIdx.x+threadIdx.x;
    C[i]=A[i]+B[i];
}

int main(){
    struct timeval start,end;
    gettimeofday(&start,NULL);
    float *A,*B,*C,*Ad,*Bd,*Cd;
    int n=1024*1024;
    int size=n*sizeof(float);
    A=(float*)malloc(size);
    B=(float*)malloc(size);
    C=(float*)malloc(size);
    for(int i=0;i<n;i++){
        A[i]=10.0;
        B[i]=90.0;
    }
    hipMalloc((void **)&Ad,size);
    hipMalloc((void **)&Bd,size);
    hipMalloc((void **)&Cd,size);

    hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
    hipMemcpy(Bd,B,size,hipMemcpyHostToDevice);
    //cudaMemcpy(Cd,C,size,cudaMemcpyHostToDevice);

    dim3 dimBlock(1024);
    dim3 dimGrid(n/1024);
    Plus<<<dimGrid,dimBlock>>>(Ad,Bd,Cd,n);

    hipMemcpy(C,Cd,size,hipMemcpyDeviceToHost);
    float max_error=0.0;
    for(int i=0;i<n;i++){
        max_error+=fabs(100.0-C[i]);
    }
    cout<<"max error is: "<<max_error<<endl;
    free(A);
    free(B);
    free(C);
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
    gettimeofday(&end,NULL);
    int timeuse=1000000*(end.tv_sec-start.tv_sec)+end.tv_usec-start.tv_usec;
    cout<<"Total time is: "<<timeuse/1000<<"ms"<<endl;
    return 0;
}





